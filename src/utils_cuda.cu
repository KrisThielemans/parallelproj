/**
 * @file utils_cuda.cu
 */

#include<stdio.h>
#include<stdlib.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>



/** @brief CUDA kernel to add array b to array a
 * 
 *  @param a first array of length n
 *  @param b first array of length n
 *  @param n length of vectors
 *
*/ 
extern "C" __global__ void add_to_first_kernel(float* a, float* b, unsigned long long n)
{
// add a vector b onto a vector a both of length n

  unsigned long long i = blockDim.x * blockIdx.x + threadIdx.x;

  if(i < n)
  {
    a[i] += b[i];
  }
}




extern "C" void CreateTextureInterp(float* imagedata, int* img_dim, hipArray** d_cuArrTex, hipTextureObject_t *texImage, bool allocate, int num_devices)
{
    //const cudaExtent extent = make_cudaExtent(img_dim[0], img_dim[1], img_dim[2]);
    const hipExtent extent = make_hipExtent(img_dim[2], img_dim[1], img_dim[0]);
    if(allocate){
        
        for (int dev = 0; dev < num_devices; dev++){
            hipSetDevice(dev);
            
            //cudaArray Descriptor
            
            hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
            //cuda Array
            hipMalloc3DArray(&d_cuArrTex[dev], &channelDesc, extent);
            //cudaCheckErrors("Texture memory allocation fail");
        }
        
    }
    for (int dev = 0; dev < num_devices; dev++){
        hipMemcpy3DParms copyParams = {0};
        hipSetDevice(dev);
        //Array creation
        copyParams.srcPtr   = make_hipPitchedPtr((void *)imagedata, extent.width*sizeof(float), extent.width, extent.height);
        copyParams.dstArray = d_cuArrTex[dev];
        copyParams.extent   = extent;
        copyParams.kind     = hipMemcpyHostToDevice;
        hipMemcpy3DAsync(&copyParams);
        //cudaCheckErrors("Texture memory data copy fail");
        //Array creation End
    }
    for (int dev = 0; dev < num_devices; dev++){
        hipSetDevice(dev);
        hipResourceDesc    texRes;
        memset(&texRes, 0, sizeof(hipResourceDesc));
        texRes.resType = hipResourceTypeArray;
        texRes.res.array.array  = d_cuArrTex[dev];
        hipTextureDesc     texDescr;
        memset(&texDescr, 0, sizeof(hipTextureDesc));
        texDescr.normalizedCoords = false;
        texDescr.filterMode = hipFilterModeLinear;
        texDescr.addressMode[0] = hipAddressModeBorder;
        texDescr.addressMode[1] = hipAddressModeBorder;
        texDescr.addressMode[2] = hipAddressModeBorder;
        texDescr.readMode = hipReadModeElementType;
        hipCreateTextureObject(&texImage[dev], &texRes, &texDescr, NULL);
        //cudaCheckErrors("Texture object creation fail");
    }
}
