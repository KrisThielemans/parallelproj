/**
 * @file joseph3d_lm_cuda.cu
 */

#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>


/** @brief 3D listmode non-tof joseph forward projector CUDA kernel
 *
 *  @param xstart array of shape [3*nlors] with the coordinates of the start points of the LORs.
 *                The start coordinates of the n-th LOR are at xstart[n*3 + i] with i = 0,1,2 
 *  @param xend   array of shape [3*nlors] with the coordinates of the end   points of the LORs.
 *                The start coordinates of the n-th LOR are at xstart[n*3 + i] with i = 0,1,2 
 *  @param img    array of shape [n0*n1*n2] containing the 3D image to be projected.
 *                The pixel [i,j,k] ist stored at [n1*n2+i + n2*k + j].
 *  @param img_origin  array [x0_0,x0_1,x0_2] of coordinates of the center of the [0,0,0] voxel
 *  @param voxsize     array [vs0, vs1, vs2] of the voxel sizes
 *  @param p           array of length nlors (output) used to store the projections
 *  @param nlors       number of projections (length of p array)
 *  @param img_dim     array with dimensions of image [n0,n1,n2]
 */
__global__ void joseph3d_lm_cuda_kernel(float *xstart, 
                                        float *xend, 
                                        float *img,
                                        float *img_origin, 
                                        float *voxsize, 
                                        float *p,
                                        unsigned long long nlors, 
                                        unsigned int *img_dim)
{
  unsigned long long i = blockDim.x * blockIdx.x + threadIdx.x;

  unsigned int n0 = img_dim[0];
  unsigned int n1 = img_dim[1];
  unsigned int n2 = img_dim[2];

  float d0, d1, d2, d0_sq, d1_sq, d2_sq; 
  float lsq, cos0_sq, cos1_sq, cos2_sq;
  unsigned short direction; 
  unsigned int i0, i1, i2;
  int i0_floor, i1_floor, i2_floor;
  int i0_ceil, i1_ceil, i2_ceil;
  float x_pr0, x_pr1, x_pr2;
  float tmp_0, tmp_1, tmp_2;

  if(i < nlors)
  {
    
    // initialize projected value to 0 
    p[i] = 0;

    // test whether the ray between the two detectors is most parallel
    // with the 0, 1, or 2 axis
    d0 = xend[i*3 + 0] - xstart[i*3 + 0];
    d1 = xend[i*3 + 1] - xstart[i*3 + 1];
    d2 = xend[i*3 + 2] - xstart[i*3 + 2];

    d0_sq = d0*d0;
    d1_sq = d1*d1;
    d2_sq = d2*d2;

    lsq = d0_sq + d1_sq + d2_sq;

    cos0_sq = d0_sq / lsq;
    cos1_sq = d1_sq / lsq;
    cos2_sq = d2_sq / lsq;

    direction = 0;
    if ((cos1_sq >= cos0_sq) && (cos1_sq >= cos2_sq))
    {
      direction = 1;
    }
    else
    {
      if ((cos2_sq >= cos0_sq) && (cos2_sq >= cos1_sq))
      {
        direction = 2;
      }
    }
 
    if (direction == 0)
    {
      // case where ray is most parallel to the 0 axis
      // we step through the volume along the 0 direction
      for(i0 = 0; i0 < n0; i0++)
      {
        // get the indices where the ray intersects the image plane
        x_pr1 = xstart[i*3 + 1] + (img_origin[direction] + i0*voxsize[direction] - 
                                   xstart[i*3 + direction])*d1 / d0;
        x_pr2 = xstart[i*3 + 2] + (img_origin[direction] + i0*voxsize[direction] - 
                                   xstart[i*3 + direction])*d2 / d0;
  
        i1_floor = (int)floor((x_pr1 - img_origin[1])/voxsize[1]);
        i1_ceil  = i1_floor + 1;
  
        i2_floor = (int)floor((x_pr2 - img_origin[2])/voxsize[2]);
        i2_ceil  = i2_floor + 1; 
  
        // calculate the distances to the floor normalized to [0,1]
        // for the bilinear interpolation
        tmp_1 = (x_pr1 - (i1_floor*voxsize[1] + img_origin[1])) / voxsize[1];
        tmp_2 = (x_pr2 - (i2_floor*voxsize[2] + img_origin[2])) / voxsize[2];

        // do bilinear interpolation 
        if ((i1_floor >= 0) && (i1_floor < n1) && (i2_floor >= 0) && (i2_floor < n2))
        {
          p[i] += img[n1*n2*i0 + n2*i1_floor + i2_floor] * (1 - tmp_1) * (1 - tmp_2);
        }
        if ((i1_ceil >= 0) && (i1_ceil < n1) && (i2_floor >= 0) && (i2_floor < n2))
        {
          p[i] += img[n1*n2*i0 + n2*i1_ceil + i2_floor] * tmp_1 * (1 - tmp_2);
        }
        if ((i1_floor >= 0) && (i1_floor < n1) && (i2_ceil >= 0) && (i2_ceil < n2))
        {
          p[i] += img[n1*n2*i0 + n2*i1_floor + i2_ceil] * (1 - tmp_1) * tmp_2;
        }
        if ((i1_ceil >= 0) && (i1_ceil < n1) && (i2_ceil >= 0) && (i2_ceil < n2))
        {
          p[i] += img[n1*n2*i0 + n2*i1_ceil + i2_ceil] * tmp_1 * tmp_2;
        }
      }
      // correct for |cos(theta)| 
      p[i] /= sqrt(cos0_sq);
    }

    //--------------------------------------------------------------------------------- 
    if (direction == 1)
    {
      // case where ray is most parallel to the 1 axis
      // we step through the volume along the 1 direction
      for (i1 = 0; i1 < n1; i1++)
      {
        // get the indices where the ray intersects the image plane
        x_pr0 = xstart[i*3 + 0] + (img_origin[direction] + i1*voxsize[direction] - 
                                   xstart[i*3 + direction])*d0 / d1;
        x_pr2 = xstart[i*3 + 2] + (img_origin[direction] + i1*voxsize[direction] - 
                                   xstart[i*3 + direction])*d2 / d1;
  
        i0_floor = (int)floor((x_pr0 - img_origin[0])/voxsize[0]);
        i0_ceil  = i0_floor + 1; 
  
        i2_floor = (int)floor((x_pr2 - img_origin[2])/voxsize[2]);
        i2_ceil  = i2_floor + 1;
  
        // calculate the distances to the floor normalized to [0,1]
        // for the bilinear interpolation
        tmp_0 = (x_pr0 - (i0_floor*voxsize[0] + img_origin[0])) / voxsize[0];
        tmp_2 = (x_pr2 - (i2_floor*voxsize[2] + img_origin[2])) / voxsize[2];
  
        if ((i0_floor >= 0) && (i0_floor < n0) && (i2_floor >= 0) && (i2_floor < n2))
        {
          p[i] += img[n1*n2*i0_floor +  n2*i1 + i2_floor] * (1 - tmp_0) * (1 - tmp_2);
        }
        if ((i0_ceil >= 0) && (i0_ceil < n0) && (i2_floor >= 0) && (i2_floor < n2))
        {
          p[i] += img[n1*n2*i0_ceil + n2*i1 + i2_floor] * tmp_0 * (1 - tmp_2);
        }
        if ((i0_floor >= 0) && (i0_floor < n0) && (i2_ceil >= 0) && (i2_ceil < n2))
        {
          p[i] += img[n1*n2*i0_floor + n2*i1 + i2_ceil] * (1 - tmp_0) * tmp_2;
        }
        if ((i0_ceil >= 0) && (i0_ceil < n0) && (i2_ceil >= 0) && (i2_ceil < n2))
        {
          p[i] += img[n1*n2*i0_ceil + n2*i1 + i2_ceil] * tmp_0 * tmp_2;
        }
      }
      // correct for |cos(theta)| 
      p[i] /= sqrt(cos1_sq);
    }

    //--------------------------------------------------------------------------------- 
    if (direction == 2)
    {
      // case where ray is most parallel to the 2 axis
      // we step through the volume along the 2 direction

      for(i2 = 0; i2 < n2; i2++)
      {
        // get the indices where the ray intersects the image plane
        x_pr0 = xstart[i*3 + 0] + (img_origin[direction] + i2*voxsize[direction] - 
                                   xstart[i*3 + direction])*d0 / d2;
        x_pr1 = xstart[i*3 + 1] + (img_origin[direction] + i2*voxsize[direction] - 
                                   xstart[i*3 + direction])*d1 / d2;
  
        i0_floor = (int)floor((x_pr0 - img_origin[0])/voxsize[0]);
        i0_ceil  = i0_floor + 1;
  
        i1_floor = (int)floor((x_pr1 - img_origin[1])/voxsize[1]);
        i1_ceil  = i1_floor + 1; 
  
        // calculate the distances to the floor normalized to [0,1]
        // for the bilinear interpolation
        tmp_0 = (x_pr0 - (i0_floor*voxsize[0] + img_origin[0])) / voxsize[0];
        tmp_1 = (x_pr1 - (i1_floor*voxsize[1] + img_origin[1])) / voxsize[1];
  
        if ((i0_floor >= 0) && (i0_floor < n0) && (i1_floor >= 0) && (i1_floor < n1))
        {
          p[i] += img[n1*n2*i0_floor + n2*i1_floor + i2] * (1 - tmp_0) * (1 - tmp_1);
        }
        if ((i0_ceil >= 0) && (i0_ceil < n0) && (i1_floor >= 0) && (i1_floor < n1))
        {
          p[i] += img[n1*n2*i0_ceil + n2*i1_floor + i2] * tmp_0 * (1 - tmp_1);
        }
        if ((i0_floor >= 0) && (i0_floor < n0) && (i1_ceil >= 0) & (i1_ceil < n1))
        {
          p[i] += img[n1*n2*i0_floor + n2*i1_ceil + i2] * (1 - tmp_0) * tmp_1;
        }
        if ((i0_ceil >= 0) && (i0_ceil < n0) && (i1_ceil >= 0) && (i1_ceil < n1))
        {
          p[i] += img[n1*n2*i0_ceil + n2*i1_ceil + i2] * tmp_0 * tmp_1;
        }
      }
      // correct for |cos(theta)| 
      p[i] /= sqrt(cos2_sq);
    }
    // correct for the voxsize
    p[i] *= voxsize[direction];
  }
}


//------------------------------------------------------------------------------------------
//------------------------------------------------------------------------------------------
//------------------------------------------------------------------------------------------

/** @brief 3D listmode non-tof joseph forward projector CUDA wrapper
 *
 *  @param h_xstart array of shape [3*nlors] with the coordinates of the start points of the LORs.
 *                  The start coordinates of the n-th LOR are at xstart[n*3 + i] with i = 0,1,2 
 *  @param h_xend   array of shape [3*nlors] with the coordinates of the end   points of the LORs.
 *                  The start coordinates of the n-th LOR are at xstart[n*3 + i] with i = 0,1,2 
 *  @param h_img    array of shape [n0*n1*n2] containing the 3D image to be projected.
 *                  The pixel [i,j,k] ist stored at [n1*n2+i + n2*k + j].
 *  @param h_img_origin  array [x0_0,x0_1,x0_2] of coordinates of the center of the [0,0,0] voxel
 *  @param h_voxsize     array [vs0, vs1, vs2] of the voxel sizes
 *  @param h_p           array of length nlors (output) used to store the projections
 *  @param nlors            number of projections (length of p array)
 *  @param h_img_dim     array with dimensions of image [n0,n1,n2]
 *  @param threadsperblock number of threads per block
 *  @param num_devices     number of CUDA devices to use. if set to -1 cudaGetDeviceCount() is used
 */
extern "C" void joseph3d_lm_cuda(float *h_xstart, 
                                 float *h_xend, 
                                 float *h_img,
                                 float *h_img_origin, 
                                 float *h_voxsize, 
                                 float *h_p,
                                 unsigned long long nlors, 
                                 unsigned int *h_img_dim,
                                 unsigned int threadsperblock,
                                 int num_devices)
{
  unsigned int blockspergrid;

  dim3 block(threadsperblock);

  // offset for chunk of projections passed to a device 
  unsigned long long dev_offset;
  // number of projections to be calculated on a device
  unsigned long long dev_nlors;

  unsigned int n0 = h_img_dim[0];
  unsigned int n1 = h_img_dim[1];
  unsigned int n2 = h_img_dim[2];

  unsigned long long img_bytes = (n0*n1*n2)*sizeof(float);
  unsigned long long proj_bytes_dev;

  // get number of avilable CUDA devices specified as <=0 in input
  if(num_devices <= 0){hipGetDeviceCount(&num_devices);}  

  // init the dynamic array of device arrays
  float **d_p              = new float * [num_devices];
  float **d_xstart         = new float * [num_devices];
  float **d_xend           = new float * [num_devices];
  float **d_img            = new float * [num_devices];
  float **d_img_origin     = new float * [num_devices];
  float **d_voxsize        = new float * [num_devices];
  unsigned int **d_img_dim = new unsigned int * [num_devices];

  printf("\n # CUDA devices: %d \n", num_devices);

  // we split the projections across all CUDA devices
  for (unsigned int i_dev = 0; i_dev < num_devices; i_dev++) 
  {
    hipSetDevice(i_dev);
    // () are important in integer division!
    dev_offset = i_dev*(nlors/num_devices);
 
    // calculate the number of projections for a device (last chunck can be a bit bigger) 
    dev_nlors = i_dev == (num_devices - 1) ? (nlors - dev_offset) : (nlors/num_devices);

    // calculate the number of bytes for the projection array on the device
    proj_bytes_dev = dev_nlors*sizeof(float);

    // calculate the number of blocks needed for every device (chunk)
    blockspergrid = (unsigned int)ceil((float)dev_nlors / threadsperblock);
    dim3 grid(blockspergrid);

    // allocate the memory for the array containing the projection on the device
    hipMalloc(&d_p[i_dev], proj_bytes_dev);
    hipMemsetAsync(d_p[i_dev], 0, proj_bytes_dev);

    hipMalloc(&d_xstart[i_dev], 3*proj_bytes_dev);
    hipMemcpyAsync(d_xstart[i_dev], h_xstart + 3*dev_offset, 3*proj_bytes_dev, 
                    hipMemcpyHostToDevice);

    hipMalloc(&d_xend[i_dev], 3*proj_bytes_dev);
    hipMemcpyAsync(d_xend[i_dev], h_xend + 3*dev_offset, 3*proj_bytes_dev, 
                    hipMemcpyHostToDevice);
   
    hipMalloc(&d_img[i_dev], img_bytes);
    hipMemcpyAsync(d_img[i_dev], h_img, img_bytes, hipMemcpyHostToDevice);

    hipMalloc(&d_img_origin[i_dev], 3*sizeof(float));
    hipMemcpyAsync(d_img_origin[i_dev], h_img_origin, 3*sizeof(float), 
                    hipMemcpyHostToDevice);

    hipMalloc(&d_voxsize[i_dev], 3*sizeof(float));
    hipMemcpyAsync(d_voxsize[i_dev], h_voxsize, 3*sizeof(float), hipMemcpyHostToDevice);

    hipMalloc(&d_img_dim[i_dev], 3*sizeof(unsigned int));
    hipMemcpyAsync(d_img_dim[i_dev], h_img_dim, 3*sizeof(unsigned int), hipMemcpyHostToDevice);


    // call the kernel
    joseph3d_lm_cuda_kernel<<<grid,block>>>(d_xstart[i_dev], d_xend[i_dev], d_img[i_dev], 
                                            d_img_origin[i_dev], d_voxsize[i_dev], 
                                            d_p[i_dev], dev_nlors, d_img_dim[i_dev]); 

    // copy projection back from device to host
    hipMemcpyAsync(h_p + dev_offset, d_p[i_dev], proj_bytes_dev, hipMemcpyDeviceToHost);

    // deallocate memory on device
    hipFree(d_p[i_dev]);
    hipFree(d_xstart[i_dev]);
    hipFree(d_xend[i_dev]);
    hipFree(d_img);
    hipFree(d_img_origin);
    hipFree(d_voxsize);
  }

  // make sure that all devices are done before leaving
  for (unsigned int i_dev = 0; i_dev < num_devices; i_dev++){hipDeviceSynchronize();}
}
